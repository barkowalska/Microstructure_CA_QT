#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "hip/hip_runtime.h"
#include ""
#include<iostream>

__constant__ int bc;
__constant__ int neighbourhood[32];
__constant__ int dimY;
__constant__ int dimX;
__constant__ int NeighbourhoodSize;

int* allocateMatrix(Domain& domain)
{
	int* matrixD;
	hipMalloc(&matrixD, sizeof(int) * domain.getDimX() * domain.getDimY());
	hipMemcpy(matrixD, domain.getMatrix(), sizeof(int) * domain.getDimY() * domain.getDimX(), hipMemcpyHostToDevice);
	return matrixD;
}

void setConstantCache(Domain& domain)
{
	int dDimY = domain.getDimY();
	int dDimX = domain.getDimX();
	int nSize = domain.getNeighbourhoodSize();
	int BC = domain.getBc();

	hipMemcpyToSymbol(HIP_SYMBOL(dimY), &dDimY, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(dimX), &dDimX, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(NeighbourhoodSize), &nSize, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(neighbourhood), domain.getNeighbourhood(), sizeof(int) * nSize);
	hipMemcpyToSymbol(HIP_SYMBOL(bc), &BC, sizeof(int));
	
}

void __global__ growGrain(int* matrixInput, int* matrixOutput)
{
	__shared__ int local_matrix[32][32];
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x  >= dimX || y >= dimY) return;

	local_matrix[threadIdx.y][threadIdx.x] = matrixInput[y * dimX + x];
	for (int k = 0; k < NeighbourhoodSize; k+=2)
	{	
		if ((x + neighbourhood[k]) >= dimX || (y + neighbourhood[k + 1]) >= dimY || (x + neighbourhood[k]) < 0 || (y + neighbourhood[k + 1]) < 0)
		{
			switch (bc) {
			case periodic:
			{
				int nX;
				int nY;
				if ((x + neighbourhood[k]) >= dimX)
				{
					nX = 0;
				}
				if ((y + neighbourhood[k + 1]) >= dimY)
				{
					nY = 0;
				}
				if ((x + neighbourhood[k]) < 0)
				{
					nX = dimX - 1;
				}
				if ((y + neighbourhood[k + 1]) < 0)
				{
					nY = dimY - 1;
				}
				local_matrix[threadIdx.y][threadIdx.x] = local_matrix[threadIdx.y][threadIdx.x] + matrixInput[dimX * nY + nX] * (local_matrix[threadIdx.y][threadIdx.x] == 0);
				break;
			}
			case absorbtion:
				
				break;
			

		}
		else local_matrix[threadIdx.y][threadIdx.x] = 
			local_matrix[threadIdx.y][threadIdx.x] + (local_matrix[threadIdx.y][threadIdx.x] == 0) * matrixInput[x + neighbourhood[k] + (y + neighbourhood[k + 1]) * dimX];
	}
	matrixOutput[y * dimX + x] = local_matrix[threadIdx.y][threadIdx.x];


}



void run(int* matrixInput, int* matrixOutput, Domain &domain, int iteracions)
{
	dim3 blockSize = { 32, 32, 1 };
	dim3 gridSize = { (domain.getDimX() + blockSize.x - 1) / blockSize.x, (domain.getDimY() + blockSize.y - 1) / blockSize.y, 1 };
	std::cout << gridSize.x << ' ' << gridSize.y << ' ' << gridSize.z << std::endl;
	for (int i = 0; i < iteracions; i++)
	{
		growGrain<<<gridSize, blockSize>>>(matrixInput, matrixOutput);
		hipDeviceSynchronize();
		
		int* tmp = matrixInput;
		matrixInput = matrixOutput;
		matrixOutput = tmp;
	}

	hipMemcpy(domain.getMatrix(),matrixInput , sizeof(int) * domain.getDimY() * domain.getDimX(), hipMemcpyDeviceToHost);

	hipFree(matrixInput);
	hipFree(matrixOutput);
}